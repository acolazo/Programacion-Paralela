
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <math.h>

#define SIZE 10000
#define THREADS 128 //best value = 256
#define PRINT 1
#define CHECK 1
#define DATATYPE struct number

/* Function declarations */
void getGridComposition(int, unsigned int*, unsigned int*);

/* Struct for not losing the global index */
struct number{
    int value;
    unsigned int index;
};

/* Warp reduction */
//template <unsigned int blockSize>
__device__ void warpReduce(DATATYPE* sdata, unsigned int tid, unsigned int i, int size){
    if ((i + 32) < size) sdata[tid] = sdata[tid].value > sdata[tid + 32].value ? sdata[tid] : sdata[tid + 32];
    if ((i + 16) < size) sdata[tid] = sdata[tid].value > sdata[tid + 16].value ? sdata[tid] : sdata[tid + 16];
    if ((i + 8) < size)  sdata[tid] = sdata[tid].value > sdata[tid + 8].value ? sdata[tid] : sdata[tid + 8];
    if ((i + 4) < size)  sdata[tid] = sdata[tid].value > sdata[tid + 4].value ? sdata[tid] : sdata[tid + 4];
    if ((i + 2) < size)  sdata[tid] = sdata[tid].value > sdata[tid + 2].value ? sdata[tid] : sdata[tid + 2];
    if ((i + 1) < size)  sdata[tid] = sdata[tid].value > sdata[tid + 1].value ? sdata[tid] : sdata[tid + 1];
    /*
    if (blockSize >=64) sdata[tid] = sdata[tid].value > sdata[tid + 32].value ? sdata[tid] : sdata[tid + 32];
    if (blockSize >=32) sdata[tid] = sdata[tid].value > sdata[tid + 16].value ? sdata[tid] : sdata[tid + 16];
    if (blockSize >=16) sdata[tid] = sdata[tid].value > sdata[tid + 8].value ? sdata[tid] : sdata[tid + 8];
    if (blockSize >=8)  sdata[tid] = sdata[tid].value > sdata[tid + 4].value ? sdata[tid] : sdata[tid + 4];
    if (blockSize >=4)  sdata[tid] = sdata[tid].value > sdata[tid + 2].value ? sdata[tid] : sdata[tid + 2];
    if (blockSize >=2)  sdata[tid] = sdata[tid].value > sdata[tid + 1].value ? sdata[tid] : sdata[tid + 1];
    */
};

/* Kernel reduction at block level */
__global__ void reduceKernel(int size, DATATYPE *g_input, DATATYPE *g_output)
{
    extern __shared__ DATATYPE sdata[];

    unsigned int tid = threadIdx.x;
  //  unsigned int gid = (blockIdx.x*blockDim.x) + tid;
    unsigned int i = (blockIdx.x*blockDim.x * 2) + tid;
    if((i + blockDim.x )< size)
        sdata[tid] = g_input[i].value > g_input[i + blockDim.x].value ? g_input[i] : g_input[ i + blockDim.x];  
    else if (i<size)
        sdata[tid] = g_input[i];

    __syncthreads();

    //do reduction in shared mem
    unsigned int s;
    for(s=blockDim.x/2; s > 32; s/=2){
        if((i + s)<size && tid < s){
            sdata[tid] = sdata[tid].value > sdata[tid + s].value ? sdata[tid] : sdata[tid + s];
        }
        __syncthreads();
    }
    
    /* Unrolling the last 6 iterations */
    unsigned int blockSize = blockDim.x;
    if(tid < 32) warpReduce(sdata, tid, i, size);
    
    /*
    if (tid < 32) {
        if ((i + s) < size){
            warpReduce<blockSize>(sdata, tid);
        }
        else{
            for(s=32; s > 0; s/=2){
                if((i + s)<size && tid < s){
                    sdata[tid] = sdata[tid].value > sdata[tid + s].value ? sdata[tid] : sdata[tid + s];
                }
                __syncthreads();
            }
        }
    } 
    */

    //write result for this block to global mem
    if (tid == 0) g_output[blockIdx.x] = sdata[tid];   
}

/* This function swaps the MAX element [which should be in position 0] with the last element of the list */
/* WARNING: This function must be called by only one block */
__global__ void swapKernel(int size, DATATYPE *g_list, DATATYPE *g_max){
    DATATYPE max;
    DATATYPE last_element;
    int index;
    unsigned int tid = threadIdx.x;

    if(tid == 0){
        max = g_max[0];
        index = max.index;
        max.index = size-1;

        last_element = g_list[size-1];
        last_element.index = index;

        g_list[index] = last_element; /* Donde estaba el valor maximo, pongo el ultimo elemento de la lista */
        g_list[size-1] = max;
    }
}


/* Wraps Kernel Calls */
void sortBySelection(int size, DATATYPE *g_list, DATATYPE *g_temp){
    unsigned int threads, blocks;
    DATATYPE *g_input;
    int N;
    double temp;
    dim3 dimGrid(1, 1, 1);
    dim3 dimBlock(1, 1, 1);

    N = size;
    getGridComposition(N, &blocks, &threads);
    dimGrid.x = blocks;
    dimBlock.x = threads;

    g_input = g_list;
    //printf("Iteracion %d\n", SIZE - size + 1);
    while(dimGrid.x > 0){
        reduceKernel<<<dimGrid, dimBlock, N * sizeof(DATATYPE)>>>(N, g_input, g_temp);

       
        //printf("Grid: %d, N: %d\n", dimGrid.x, N);
        
        temp = (double) N / (dimBlock.x * 2);
        N = ceil(temp);

        //dimGrid.x>>=1;
        dimGrid.x = (dimGrid.x > dimBlock.x) || (dimGrid.x == 1) ? (dimGrid.x / dimBlock.x) : 1;

        g_input = g_temp;
    }
    swapKernel<<<1, 1>>>(size, g_list, g_temp);

    if(size > 2){
        sortBySelection(size-1, g_list, g_temp);
    }
    

    return;
}

/* Get the number of blocks and threads per block */
void getGridComposition(int size, unsigned int* blocks, unsigned int* threads){

    *threads = THREADS;
    *blocks = 1;

    while (((*blocks) * (*threads) * 2) < size){
        *blocks<<=1;
    }

    return;
}

/* Print list */
void printResults(DATATYPE *sorted_list)
{
    for (int i = 0; i < SIZE; i++)
    {
        printf("%d\n", sorted_list[i].value);
    }
    printf("\n");
    return;
}

/* Check if results are correct */
int checkResults(DATATYPE *sorted_list){
    unsigned int check = 1;
    for (unsigned int i = 1; i < (SIZE + 1); i++)
    {
        if(sorted_list[i-1].value != i)
            check = 0;
    }

    if(check)
        printf("Resultados correctos!\n");
    else
        printf("Resultados incorrectos!\n");

    return check;
    
}


int main(void)
{
    DATATYPE *list, *list_g, *list_g_o;

    srand(time(NULL));

    list = (DATATYPE *)malloc(SIZE * sizeof(DATATYPE));
    if(list == NULL){
        printf("Error alocando memoria.\n");
        exit(0);
    }

    hipMalloc((void**)&list_g, SIZE * sizeof(DATATYPE));
    hipMalloc((void**)&list_g_o, SIZE * sizeof(DATATYPE));

    for (int i = 0; i < SIZE; i++)
    {
        //list[i] = rand() % 100;
        list[i].value = SIZE - i;
        list[i].index = i;
    }

    hipMemcpy(list_g, list, SIZE * sizeof(DATATYPE), hipMemcpyHostToDevice);

    // Perform Reduce Operation
    sortBySelection(SIZE, list_g, list_g_o);

    hipMemcpy(list,list_g , SIZE * sizeof(DATATYPE), hipMemcpyDeviceToHost);

    if (PRINT)
    {
        printResults(list);
    }

    if(CHECK)
    {
        checkResults(list);
    }
    
    hipFree(list_g);
    hipFree(list_g_o);
    free(list);
}