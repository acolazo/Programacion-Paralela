
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <math.h>

#define SIZE 8 * 1000
#define maxSharedMemory 49152 //bytes
#define THREADS 256 //best value = 256

#define SORT 0
#define TestReduction 1
#define PRINT 0
#define printErrors 0
#define CHECK 1

#define DATATYPE struct number
#define VALUETYPE int
#define MINVALUE INT_MIN

#define OPTION 1
/*
1: i+1 
2: SIZE-i
3: rand() % 100
*/

#define RECORDTIME 1
//#define CUDA_ERROR_CHECK

/* Function declarations */
void getGridComposition(int, unsigned int*, unsigned int*, unsigned int);

/* Struct for not losing the global index */
struct number{
    VALUETYPE value;
    int index;
};

/* Error Checking */


#define CudaSafeCall( err ) __cudaSafeCall(err, __FILE__, __LINE__)
#define CudaCheckError() __cudaCheckError( __FILE__, __LINE__ )

inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
    #ifdef CUDA_ERROR_CHECK
    if ( cudaSuccess != err ){
        fprintf ( stderr, "cudaSafeCall() failed at %s:%i : %s\n", file, line, cudaGetErrorString(err) );
        exit(-1);

    }
    #endif
    return;
}

inline void __cudaCheckError ( const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    cudaError err = cudaGetLastError();
    if(cudaSuccess != err){
        fprintf( stderr, "cudaCheckError() failed at %s:%i : %s\n", file, line, cudaGetErrorString( err ) );
        exit(-1);
    }

    /* Can affect performance. Comment if needed. */
    err = cudaDeviceSynchronize();
    if(cudaSuccess != err){
        fprintf( stderr, "cudaCheckError() with sync failed at %s:%i : %s\n", file, line, cudaGetErrorString( err ) );
        exit(-1);
    }
    #endif
    return;
}

/* Warp reduction */
template <unsigned int blockSize>
__device__ void warpReduce(DATATYPE* sdata, unsigned int tid, unsigned int i, int size){
    if (blockSize >=64) sdata[tid] = sdata[tid].value > sdata[tid + 32].value ? sdata[tid] : sdata[tid + 32];
    if (blockSize >=32) sdata[tid] = sdata[tid].value > sdata[tid + 16].value ? sdata[tid] : sdata[tid + 16];
    if (blockSize >=16) sdata[tid] = sdata[tid].value > sdata[tid + 8].value ? sdata[tid] : sdata[tid + 8];
    if (blockSize >=8)  sdata[tid] = sdata[tid].value > sdata[tid + 4].value ? sdata[tid] : sdata[tid + 4];
    if (blockSize >=4)  sdata[tid] = sdata[tid].value > sdata[tid + 2].value ? sdata[tid] : sdata[tid + 2];
    if (blockSize >=2)  sdata[tid] = sdata[tid].value > sdata[tid + 1].value ? sdata[tid] : sdata[tid + 1];
};

/* Kernel reduction at block level */
template <unsigned int blockSize>
__global__ void reduceKernel(int size, DATATYPE *g_input, DATATYPE *g_output)
{
    extern __shared__ DATATYPE sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = (blockIdx.x*blockDim.x * 2) + tid;
    
    if((i + blockDim.x )< size)
        sdata[tid] = g_input[i].value > g_input[i + blockDim.x].value ? g_input[i] : g_input[ i + blockDim.x];  
    else if (i<size){
        sdata[tid] = g_input[i];
    }
    
    else{
        DATATYPE min_value;
        min_value.value = MINVALUE;
        sdata[tid] = min_value;
    }
    
    
        

    __syncthreads();

    /* Unrolling all iterations */
    if (blockSize >= 1024) {
        if (tid < 512) {
            sdata[tid] = sdata[tid].value > sdata[tid + 512].value ? sdata[tid] : sdata[tid + 512]; 
        }
        __syncthreads(); 
    }
    if (blockSize >= 512) {
        if (tid < 256) { 
            sdata[tid] = sdata[tid].value > sdata[tid + 256].value ? sdata[tid] : sdata[tid + 256];
        }
        __syncthreads(); 
    }
    if (blockSize >= 256) {
        if (tid < 128) { 
            sdata[tid] = sdata[tid].value > sdata[tid + 128].value ? sdata[tid] : sdata[tid + 128];
        } 
        __syncthreads(); 
    }
    if (blockSize >= 128) {
        if (tid <  64) { 
            sdata[tid] = sdata[tid].value > sdata[tid + 64].value ? sdata[tid] : sdata[tid + 64];
        }
        __syncthreads(); 
    }   
    
    if(tid < 32){
        warpReduce<blockSize>(sdata, tid, i, size);
    }

    //write result for this block to global mem
    if (tid == 0) g_output[blockIdx.x] = sdata[tid];   
}

/* This function swaps the MAX element [which should be in position 0] with the last element of the list */
/* WARNING: This function must be called by only one block */
__global__ void swapKernel(int size, DATATYPE *g_list, DATATYPE *g_max, VALUETYPE *g_output){
    DATATYPE max;
    DATATYPE last_element;
    int index;
    //unsigned int tid = threadIdx.x;

        max = g_max[0];
        index = max.index;
        max.index = size-1;

        last_element = g_list[size-1];
        last_element.index = index;

        g_list[index] = last_element; /* Donde estaba el valor maximo, pongo el ultimo elemento de la lista */
        
        g_output[size - 1] = max.value; /* Pongo el maximo en la lista de resultados */

        max.value = MINVALUE; /* Cambio el valor del maximo para que sea el minimo posible */
        g_list[size-1] = max;

}


/* Kernel call that wraps data into a struct with index */
__global__ void wrapKernel(int size, DATATYPE *g_wrapped_list, VALUETYPE * g_list ){
    unsigned int gid = threadIdx.x + blockDim.x * blockIdx.x;
    DATATYPE myData;

    if ( gid < size ){
        myData.index = gid;
        myData.value = g_list[gid];

        g_wrapped_list[gid] = myData;
    }
}

/* Kernel call that unwraps data into an array of VALUETYPE */
__global__ void unwrapKernel(int size, DATATYPE *g_wrapped_list, VALUETYPE * g_list ){
    unsigned int gid = threadIdx.x + blockDim.x * blockIdx.x;

    if ( gid < size ){
         g_list[gid] = g_wrapped_list[gid].value;
    }
}


/* Wraps Reduction Kernel Call */
int reduceMax(int size, DATATYPE *g_list, DATATYPE *g_temp, DATATYPE *g_temp_results){
    static int counter = 0;
    unsigned int threads, blocks;
    DATATYPE *g_input, *g_output, *g_iteration_list;
    int N, iterations, maxAllowedSize, CONST_N;
    unsigned int maxLoadToShared, A;
    double temp;
    dim3 dimGrid(1, 1, 1);
    dim3 dimBlock(1, 1, 1);
    

    g_iteration_list = g_list;
    /* Check if we can do one kernel call or more */
    iterations = 1;
    CONST_N = size;

    maxLoadToShared = 1;
    while(maxLoadToShared * 2 * sizeof(DATATYPE) < maxSharedMemory)
        maxLoadToShared<<=1;



    maxAllowedSize = maxLoadToShared * 2;

    while(CONST_N > maxAllowedSize ){
       CONST_N -= maxAllowedSize;
       iterations++;
    }
    
    for(int i = 0; i < iterations; i++){
        /* Get Grid Composition */
        g_input = g_iteration_list;
        N = CONST_N;
        getGridComposition(N, &blocks, &threads, 2);
        dimGrid.x = blocks;
        dimBlock.x = threads;

        //printf("N: %d, Blocks: %d, Threads: %d\n", N, blocks, threads);

        g_output = g_temp;

        /* Perform the reduction for N elements */
        while(dimGrid.x > 0){
            /* */
            A = dimGrid.x * dimBlock.x;
            /* */

            if(dimGrid.x == 1){
                g_output = g_temp_results + i;
            }
            counter++;
            switch(dimBlock.x){
                case 1024:
                    reduceKernel<1024><<<dimGrid, dimBlock, A * sizeof(DATATYPE)>>>(N, g_input, g_output);
                    break;
                case 512:
                    reduceKernel<512><<<dimGrid, dimBlock, A * sizeof(DATATYPE)>>>(N, g_input, g_output);
                    break;
                case 256:
                    reduceKernel<256><<<dimGrid, dimBlock, A * sizeof(DATATYPE)>>>(N, g_input, g_output);
                    break;
                case 128:
                    reduceKernel<128><<<dimGrid, dimBlock, A * sizeof(DATATYPE)>>>(N, g_input, g_output);
                    break;
                case 64:
                    reduceKernel<64><<<dimGrid, dimBlock, A * sizeof(DATATYPE)>>>(N, g_input, g_output);
                    break;
                case 32:
                    reduceKernel<32><<<dimGrid, dimBlock, A * sizeof(DATATYPE)>>>(N, g_input, g_output);
                    break;  
                case 16:
                    reduceKernel<16><<<dimGrid, dimBlock, A * sizeof(DATATYPE)>>>(N, g_input, g_output);
                    break;
                case 8:
                    reduceKernel<8><<<dimGrid, dimBlock, A * sizeof(DATATYPE)>>>(N, g_input, g_output);
                    break;
                case 4:
                    reduceKernel<4><<<dimGrid, dimBlock, A * sizeof(DATATYPE)>>>(N, g_input, g_output);
                    break;
                case 2:
                    reduceKernel<2><<<dimGrid, dimBlock, A * sizeof(DATATYPE)>>>(N, g_input, g_output);
                    break;
                case 1:
                    reduceKernel<1><<<dimGrid, dimBlock, A * sizeof(DATATYPE)>>>(N, g_input, g_output);
                    break;
            }
            CudaCheckError();
            
            temp = (double) N / (dimBlock.x * 2);
            if (N % (dimBlock.x * 2) != 0) temp++;
            N = temp;

            dimGrid.x = (dimGrid.x > (dimBlock.x * 2)) || (dimGrid.x == 1) ? (dimGrid.x / (dimBlock.x * 2)) : 1;

            g_input = g_temp;
        }

        g_iteration_list += CONST_N;
        CONST_N = maxAllowedSize;
    }

   
    /* Recursive call to reduce Wrapper */
    if(iterations > 1){
        reduceMax(iterations, g_temp_results, g_temp, g_temp_results);
    }
    

    return counter;
}


/* Calls the reduction wrapper and sorts the max results */
void sortBySelection(int size, DATATYPE *g_list, DATATYPE *g_temp, DATATYPE * g_temp_results, VALUETYPE * g_results){
    reduceMax(size, g_list, g_temp, g_temp_results);

    swapKernel<<<1, 1>>>(size, g_list, g_temp_results, g_results);

    if(size > 2){
        sortBySelection(size-1, g_list, g_temp, g_temp_results, g_results);
    }
    

    return;
}

/* Calls the iterative reduction wrapper and sorts the max results */
void sortBySelectionIterative(int size, DATATYPE *g_list, DATATYPE *g_temp, DATATYPE * g_temp_results, VALUETYPE * g_results){

    for(int i = size; i > 0; i--){
        reduceMax(i, g_list, g_temp, g_temp_results);
        swapKernel<<<1, 1>>>(i, g_list, g_temp_results, g_results);
    }

    

    return;
}

/* Get the number of blocks and threads per block */
void getGridComposition(int size, unsigned int* blocks, unsigned int* threads, unsigned int data_per_thread){

    *threads = THREADS;
    *blocks = 1;

    while (((*blocks) * (*threads) * data_per_thread) < size){
        *blocks<<=1;
    }

    
    /*
    if(*blocks == 1){
        while( ( *threads * data_per_thread / 2 ) > size && (*threads > 1)){
            *threads >>=1;
        }
    }
    */
    

    return;
}

void printResults(VALUETYPE *sorted_list)
{   
    if(printErrors){
        for (int i = 0; i < SIZE; i++)
        {
            if(sorted_list[i] != (i + 1)) printf("%d: %d \n", i + 1, sorted_list[i]);
        }
        printf("\n");
    }
    else{
        for (int i = 0; i < SIZE; i++)
        {
            printf("%d\n", sorted_list[i]);
        }
        printf("\n");
    }
    
    return;
}

int checkResults(VALUETYPE *sorted_list){
    unsigned int check = 1;
    unsigned int i;
    for (i = 0; i < SIZE; i++)
    {
        if(sorted_list[i] != (i + 1))
            check = 0;
    }

    if(check)
        printf("Resultados correctos!\n");
    else
        printf("Resultados incorrectos!\n");

    return check;
    
}

int main(void)
{
    DATATYPE *list_g, *list_g_o, *g_temp_results;
    VALUETYPE *list, *list_g_unwrapped;
    int allocate_exceded_share_mem;
    srand(time(NULL));

    /* Allocate Host memory */
    list = (VALUETYPE *)malloc(SIZE * sizeof(VALUETYPE));
    if(list == NULL){
        printf("Error alocando memoria.\n");
        exit(-1);
    }

    /* Allocate device memory */
    CudaSafeCall( hipMalloc((void**)&list_g_unwrapped, SIZE * sizeof(VALUETYPE)) );
    CudaSafeCall( hipMalloc((void**)&list_g, SIZE * sizeof(DATATYPE)) );
    CudaSafeCall( hipMalloc((void**)&list_g_o, maxSharedMemory / THREADS ) );

    allocate_exceded_share_mem = 1;
    for(int i = (SIZE * sizeof(DATATYPE)); i > maxSharedMemory; i-=maxSharedMemory)
        allocate_exceded_share_mem++;

    CudaSafeCall (hipMalloc((void**)&g_temp_results, allocate_exceded_share_mem * sizeof(DATATYPE)) );

    /* Initialize data */
   /* Initialize data */
   for (int i = 0; i < SIZE; i++)
   {   
       switch(OPTION){
           case 1:
           list[i] = i + 1;
           break;
           case 2:
           list[i] = SIZE - i;
           break;
           case 3:
           list[i] = rand() % 100;
           break;
       }        
   }

    /* Wrap Data into a struct with index for sorting */
    unsigned int threads, blocks;
    dim3 dimGrid(1, 1, 1);
    dim3 dimBlock(1, 1, 1);

    CudaSafeCall( hipMemcpy(list_g_unwrapped, list, SIZE * sizeof(VALUETYPE), hipMemcpyHostToDevice) );

    getGridComposition(SIZE, &blocks, &threads, 1);
    dimGrid.x = blocks;
    dimBlock.x = threads;

    wrapKernel<<<dimGrid, dimBlock>>>(SIZE, list_g, list_g_unwrapped );
    /* End of wrapping data */


    if (SORT){
        /* Record time */
        hipEvent_t start, stop;
        if(RECORDTIME){
            hipEventCreate(&start);
            hipEventCreate(&stop);
            hipEventRecord(start);
        }
        sortBySelectionIterative(SIZE, list_g, list_g_o, g_temp_results, list_g_unwrapped);
        if(RECORDTIME){
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            float milliseconds = 0;
            hipEventElapsedTime(&milliseconds, start, stop);
            printf("Pasaron %f milisegundos\n", milliseconds);
        }
        CudaSafeCall( hipMemcpy(list,list_g_unwrapped , SIZE * sizeof(VALUETYPE), hipMemcpyDeviceToHost) );
    } 

    if (TestReduction){
        /* Record time */
        int contador;
        hipEvent_t start, stop;
        if(RECORDTIME){
            hipEventCreate(&start);
            hipEventCreate(&stop);
            hipEventRecord(start);
        }
        contador = reduceMax(SIZE, list_g, list_g_o, g_temp_results);
        if(RECORDTIME){
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            float milliseconds = 0;
            hipEventElapsedTime(&milliseconds, start, stop);
            printf("Pasaron %f milisegundos\n", milliseconds);
        } 
        unwrapKernel<<<1, 1>>>(1, g_temp_results, list_g_unwrapped );
        printf("Entro %d veces a la funcion de kernel\n", contador);
        CudaSafeCall( hipMemcpy(list,list_g_unwrapped , 1 * sizeof(VALUETYPE), hipMemcpyDeviceToHost) );

    } 
    
    /* Unwrap data into an array of VALUETYPE */
    // unwrapKernel<<<dimGrid, dimBlock>>>(SIZE, list_g, list_g_unwrapped );

    /* End of unwrapping data */

    if(TestReduction){
        printf("El maximo es %d\n", list[0]);
    }

    if (PRINT && SORT)
    {
        printResults(list);
    }
    if(CHECK && SORT)
    {
        checkResults(list);
    }

    printf("Allocated Exceded Mem: %d\n", allocate_exceded_share_mem);

    CudaSafeCall ( hipFree(g_temp_results) );
    CudaSafeCall ( hipFree(list_g) );
    CudaSafeCall ( hipFree(list_g_o) );
    CudaSafeCall ( hipFree(list_g_unwrapped) );
    free(list);
}